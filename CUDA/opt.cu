#include "hip/hip_runtime.h"
#include "common.h"
#include <cstdio>
#define NUM_THREADS 256

// Put any static global variables here that you will use throughout the simulation.

int bins_per_row;
int bin_count;
double bin_size;

int* heads;
int* linked_list;

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;

    // Check if the two particles should interact
    if (r2 > cutoff * cutoff)
        return;

    r2 = fmax( r2, min_r * min_r );
    //r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    // Very simple short-range repulsive force
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

/*
       0   1
     +---X---->
   + +---+---+
0  | | 0 | 1 |
   Y +-------+
1  | | 2 | 3 |
   | +---+---+
   v
*/

__device__ int inline get_bin_id(particle_t& particle, int bins_per_row, double bin_size) {
    int x, y;
    y = int(particle.y / bin_size);
    x = int(particle.x / bin_size);
    if (x == bins_per_row) {
        x--;
    }
    if (y == bins_per_row) {
        y--;
    }
    return y * bins_per_row + x;
}


__global__ void rebin_gpu(particle_t* parts, int num_parts, int bins_per_row, int bin_count, double bin_size, int* heads, int* linked_list) {
    // Get thread (particle) ID
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < bin_count) {
        heads[index] = -1;
    }
    if (index >= num_parts)
        return;
    int bin_id = get_bin_id(parts[index], bins_per_row, bin_size);
    linked_list[index] = atomicExch(&heads[bin_id], index);
}

__device__ bool inline has_up(int bin_id, int bin_row_count) {
    return bin_id - bin_row_count > -1;
}
__device__ bool inline has_down(int bin_id, int bin_row_count, int bin_count) {
    return bin_id + bin_row_count < bin_count;
}
__device__ bool inline has_left(int bin_id, int bin_row_count) {
    return bin_id % bin_row_count != 0;
}
__device__ bool inline has_right(int bin_id, int bin_row_count) {
    return bin_id % bin_row_count != bin_row_count - 1;
}

__device__ void inline loop(particle_t* parts, int i, int another_bin_id, int* heads, int* linked_list) {
    int ptr = heads[another_bin_id];
    for (; ptr != -1; ptr = linked_list[ptr]) {
        apply_force_gpu(parts[i], parts[ptr]);
    }
}

__global__ void compute_forces_gpu(particle_t* parts, int num_parts, int bins_per_row, int bin_count, double bin_size, int* heads, int* linked_list) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    parts[tid].ax = parts[tid].ay = 0;
    int bin_id = get_bin_id(parts[tid], bins_per_row, bin_size);

    // self
    loop(parts, tid, bin_id, heads, linked_list);

    // up
    if (has_up(bin_id, bins_per_row)) {
        loop(parts, tid, bin_id - bins_per_row, heads, linked_list);
    }
    // up right
    if (has_up(bin_id, bins_per_row) && has_right(bin_id, bins_per_row)) {
        loop(parts, tid, bin_id - bins_per_row + 1, heads, linked_list);
    }
    // right
    if (has_right(bin_id, bins_per_row)) {
        loop(parts, tid, bin_id + 1, heads, linked_list);
    }
    // down right
    if (has_down(bin_id, bins_per_row, bin_count) && has_right(bin_id, bins_per_row)) {
        loop(parts, tid, bin_id + bins_per_row + 1, heads, linked_list);
    }
    // down
    if (has_down(bin_id, bins_per_row, bin_count)) {
        loop(parts, tid, bin_id + bins_per_row, heads, linked_list);
    }
    // down left
    if (has_down(bin_id, bins_per_row, bin_count) && has_left(bin_id, bins_per_row)) {
        loop(parts, tid, bin_id + bins_per_row - 1, heads, linked_list);
    }
    // left
    if (has_left(bin_id, bins_per_row)) {
        loop(parts, tid, bin_id - 1, heads, linked_list);
    }
    // up left
    if (has_up(bin_id, bins_per_row) && has_left(bin_id, bins_per_row)) {
        loop(parts, tid, bin_id - bins_per_row - 1, heads, linked_list);
    }
}

// Integrate the ODE
__global__ void move_gpu(particle_t* parts, int num_parts, double size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &parts[tid];

    // Slightly simplified Velocity Verlet integration
    // Conserves energy better than explicit Euler method
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    // Bounce from walls
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here

    bins_per_row = size / cutoff;
    bin_count = bins_per_row * bins_per_row;
    bin_size = size / bins_per_row;

    hipMalloc((void**)&heads, bin_count * sizeof(int));
    hipMalloc((void**)&linked_list, num_parts * sizeof(int));

    int blockSize = NUM_THREADS;
    int numBlocks = (bin_count + blockSize - 1) / blockSize;
    rebin_gpu<<<numBlocks, blockSize>>>(parts, num_parts, bins_per_row, bin_count, bin_size, heads, linked_list);
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    int blockSize = NUM_THREADS;
    int numBlocks;
    numBlocks = (num_parts + blockSize - 1) / blockSize;

    // Compute forces
    compute_forces_gpu<<<numBlocks, blockSize>>>(parts, num_parts, bins_per_row, bin_count, bin_size, heads, linked_list);
    hipDeviceSynchronize();

    // Move particles
    move_gpu<<<numBlocks, blockSize>>>(parts, num_parts, size);
    hipDeviceSynchronize();

    numBlocks= (bin_count + blockSize - 1) / blockSize;
    rebin_gpu<<<numBlocks, blockSize>>>(parts, num_parts, bins_per_row, bin_count, bin_size, heads, linked_list);
    hipDeviceSynchronize();
}